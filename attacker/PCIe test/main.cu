
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

int main(void)
{
  int N = 20 * (1 << 20);
  float *x, *y, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEvent_t startH2D, stopH2D;
  hipEventCreate(&startH2D);
  hipEventCreate(&stopH2D);

  hipEvent_t startD2H, stopD2H;
  hipEventCreate(&startD2H);
  hipEventCreate(&stopD2H);
  
  // host2device
  hipEventRecord(startH2D);
  hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

  hipEventRecord(stopH2D);
  hipEventSynchronize(stopH2D);
  float millisecondsH2D = 0;
  hipEventElapsedTime(&millisecondsH2D, startH2D, stopH2D);
//   printf("HostToDevice Latency (ms): %f\n", millisecondsH2D);
  printf("%f,", millisecondsH2D); 
  hipEventRecord(start);

  // Perform SAXPY on 1M elements
  saxpy<<<(N+511)/512, 512>>>(N, 2.0f, d_x, d_y);

  
  //deice2host
  hipEventRecord(startD2H);

  hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

  hipEventRecord(stopD2H);
  hipEventSynchronize(stopD2H);
  float millisecondsD2H = 0;
  hipEventElapsedTime(&millisecondsD2H, startD2H, stopD2H);
//   printf("DeviceToHost Latency (ms): %f\n", millisecondsD2H);
  printf("%f,", millisecondsD2H);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  float duration = milliseconds-millisecondsD2H;
//   printf("Effective Bandwidth (GB/s): %f\n", N*4*3/duration/1e6);
  printf("%f", N*4*3/duration/1e6);
  printf("\n");
}